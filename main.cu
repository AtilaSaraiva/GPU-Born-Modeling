#include "hip/hip_runtime.h"
/*
Hello world of wave propagation in CUDA. FDTD acoustic wave propagation in homogeneous medium. Second order accurate in time and eigth in space.

Oleg Ovcharenko
Vladimir Kazei, 2019

oleg.ovcharenko@kaust.edu.sa
vladimir.kazei@kaust.edu.sa
*/

#include <rsf.hh>
#include <iostream>
#include <string>
#include "stdio.h"
#include "math.h"
#include "stdlib.h"
#include "string.h"
/*
Add this to c_cpp_properties.json if linting isn't working for CUDA libraries
"includePath": [
                "/usr/local/cuda-10.0/targets/x86_64-linux/include",
                "${workspaceFolder}/**"
            ],
*/

#include "cuwaveprop2d.cu"

using namespace std;

void modeling(int nx, int ny, int nb, int nr, int nt, int gxbeg, int gxend, int isrc, int jsrc, float dx, float dy, float dt, float *h_vpe, float *h_dvpe, float *h_tapermask, float *h_data, float *h_directwave, float * h_wavelet, bool snaps, int nshots, int incShots, sf_file Fonly_directWave, sf_file Fdata_directWave, sf_file Fdata);

void dummyVelField(int nxb, int nyb, int nb, float *h_vpe, float *h_dvpe)
{
    for (int i = 0; i < nyb; i++){
        for (int j = 0; j < nxb; j++){
            h_dvpe[j * nyb + i]  = h_vpe[j * nyb + nb];
        }
    }
}

void expand(int nb, int nyb, int nxb, int nz, int nx, float *a, float *b)
/*< expand domain of 'a' to 'b':  a, size=nz*nx; b, size=nyb*nxb;  >*/
{
    int iz,ix;
    for     (ix=0;ix<nx;ix++) {
        for (iz=0;iz<nz;iz++) {
            b[(nb+ix)*nyb+(nb+iz)] = a[ix*nz+iz];
        }
    }
    for     (ix=0; ix<nxb; ix++) {
        for (iz=0; iz<nb; iz++)         b[ix*nyb+iz] = b[ix*nyb+nb];//top
        for (iz=nz+nb; iz<nyb; iz++) b[ix*nyb+iz] = b[ix*nyb+nb+nz-1];//bottom
    }
    for (iz=0; iz<nyb; iz++){
        for(ix=0; ix<nb; ix++)  b[ix*nyb+iz] = b[nb*nyb+iz];//left
        for(ix=nb+nx; ix<nxb; ix++)     b[ix*nyb+iz] = b[(nb+nx-1)*nyb+iz];//right
    }
}

void abc_coef (int nb, float *abc)
{
    for(int i=0; i<nb; i++){
        abc[i] = exp (-pow(0.001 * (nb - i + 1),2.0));
    }
}

void taper (int nx, int ny, int nb, float *abc, float *campo)
{
    int nxb = nx + 2 * nb;
    int nyb = ny + 2 * nb;
    for(int j=0; j<nxb; j++){
        for(int i=0; i<nb; i++){
            campo[j * nyb + i] *= abc[i];
            campo[j * nyb + (nb + ny + i)] *= abc[nb - i - 1];
        }
    }
    for(int i=0; i<nyb; i++){
        for(int j=0; j<nb; j++){
            campo[j * nyb + i] *= abc[j];
            campo[(nb + nx + j) * nyb + i] *= abc[nb - j - 1];
        }
    }
}

sf_file createFile3D (const char *name, int dimensions[3], float spacings[3], int origins[3])
{
    sf_file Fdata = NULL;
    Fdata = sf_output(name);
    char key_n[6],key_d[6],key_o[6];
    for (int i = 0; i < 3; i++){
        sprintf(key_n,"n%i",i+1);
        sprintf(key_d,"d%i",i+1);
        sprintf(key_o,"o%i",i+1);
        sf_putint(Fdata,key_n,dimensions[i]);
        sf_putint(Fdata,key_d,spacings[i]);
        sf_putint(Fdata,key_o,origins[i]);
    }

    return Fdata;
}

typedef struct{
    int nShots;
    int srcPosX;
    int srcPosY;
    int firstReceptorPos;
    int nReceptors;
    int lastReceptorPos;
    int incShots;
    int modelNx;
    int modelNy;
    int modelNxBorder;
    int modelNyBorder;
    int modelDx;
    int modelDy;
    int taperBorder;
    // Auxiliaries
    size_t nxy;
    size_t nbxy;
    size_t nbytes;
} geometry;

geometry getParameters(sf_file FvelModel)
{
    geometry param;
    sf_getint("nr",&param.nReceptors);
    sf_getint("isrc",&param.srcPosY);
    sf_getint("jsrc",&param.srcPosX);
    sf_getint("gxbeg",&param.firstReceptorPos);
    sf_getint("nshots",&param.nShots);
    sf_getint("incShots",&param.incShots);
    sf_histint(FvelModel, "n1",&param.modelNy);
    sf_histint(FvelModel, "n2", &param.modelNx);
    sf_histint(FvelModel, "d1",&param.modelDy);
    sf_histint(FvelModel, "d2", &param.modelDx);
    param.lastReceptorPos = param.firstReceptorPos + param.nReceptors;
    param.taperBorder = 0.2 * param.modelNx;
    param.nxy = param.modelNx * param.modelNy;
    param.modelNxBorder = param.modelNx + 2 * param.taperBorder;
    param.modelNyBorder = param.modelNy + 2 * param.taperBorder;
    param.nbxy = param.modelNxBorder * param.modelNyBorder;
    param.nbytes = param.nbxy * sizeof(float); // bytes to store modelNxBorder * modelNyBorder
    return param;
}

void test_getParameters (geometry param)
{
    cerr<<"param.incShots: "<<param.incShots<<endl;
    cerr<<"param.modelDims[0] "<<param.modelNx<<param.modelNy<<endl;
    cerr<<"param.nShots "<<param.nShots<<endl;
    cerr<<"param.nReceptors "<<param.nReceptors<<endl;
    cerr<<"param.firstReceptorPos "<<param.firstReceptorPos<<endl;
    cerr<<"param.lastReceptorPos "<<param.lastReceptorPos<<endl;
}

typedef struct{
    float *velField;
    float *extVelField;
    float *firstLayerVelField;
    float maxVel;
} velocity;

velocity getVelFields(sf_file FvelModel, geometry param)
{
    velocity h_model;

    h_model.velField = new float[param.nxy];
    sf_floatread(h_model.velField, param.nxy, FvelModel);

    h_model.extVelField = new float[param.nbxy];
    memset(h_model.extVelField,0,param.nbytes);
    expand(param.taperBorder, param.modelNyBorder, param.modelNxBorder, param.modelNy, param.modelNx, h_model.velField, h_model.extVelField);

    h_model.maxVel = h_model.velField[0];
    for(int i=1; i < param.nxy; i++){
        if(h_model.velField[i] > h_model.maxVel){
            h_model.maxVel = h_model.velField[i];
        }
    }

    h_model.firstLayerVelField = new float[param.nbxy];
    dummyVelField(param.modelNxBorder, param.modelNyBorder, param.taperBorder, h_model.extVelField, h_model.firstLayerVelField);

    return h_model;
}

/*
===================================================================================
MAIN
===================================================================================
*/
int main(int argc, char *argv[])
{
    /* Main program that reads and writes data and read input variables */
    bool verb;
    sf_init(argc,argv); // init RSF
    if(! sf_getbool("verb",&verb)) verb=0;

    // Setting up I/O files
    sf_file Fvel=NULL;
    Fvel = sf_input("vel");

    // Getting command line parameters
    geometry param = getParameters(Fvel);

    // Allocate memory for velocity model
    velocity h_model = getVelFields (Fvel, param);

    printf("MODEL:\n");
    printf("\t%i x %i\t:param.modelNy x param.modelNx\n", param.modelNy, param.modelNx);
    printf("\t%f\t:param.modelDx\n", param.modelDx);
    printf("\t%f\t:h_model.velField[0]\n", h_model.velField[0]);

    cerr<<"vp = "<<h_model.maxVel<<endl;
    cerr<<"param.taperBorder = "<<param.taperBorder<<endl;

    // Taper mask
    float *h_abc = new float[param.taperBorder];
    float *h_tapermask = new float[nbxy];
    for(int i=0; i < nbxy; i++){
        h_tapermask[i] = 1;
    }
    abc_coef(param.taperBorder, h_abc);
    taper(param.modelNx, param.modelNy, param.taperBorder, h_abc, h_tapermask);


    // Time stepping
    float t_total = 2.5;               /* total time of wave propagation, sec */
    float dt = 0.5 * param.modelDx / h_model.maxVel;         /* time step assuming constant vp, sec */
    int nt = round(t_total / dt);      /* number of time steps */
    int snap_step = round(0.1 * nt);   /* save snapshot every ... steps */

    printf("TIME STEPPING:\n");
    printf("\t%e\t:t_total\n", t_total);
    printf("\t%e\t:dt\n", dt);
    printf("\t%i\t:nt\n", nt);

    // Data
    float *h_data = new float[param.nReceptors * nt];
    float *h_directwave = new float[param.nReceptors * nt];

    // Source
    float f0 = 10.0;                    /* source dominant frequency, Hz */
    float t0 = 1.2 / f0;                /* source padding to move wavelet from left of zero */

    float *h_wavelet, *h_time;
    float tbytes = nt * sizeof(float);
    h_time = (float *)malloc(tbytes);
    h_wavelet = (float *)malloc(tbytes);

    // Fill source waveform vector
    float a = PI * PI * f0 * f0;            /* const for wavelet */
    float dt2dx2 = (dt * dt) / (param.modelDx * param.modelDx);   /* const for fd stencil */
    for (int it = 0; it < nt; it++)
    {
        h_time[it] = it * dt;
        // Ricker wavelet (Mexican hat), second derivative of Gaussian
        h_wavelet[it] = 1e10 * (1.0 - 2.0 * a * pow(h_time[it] - t0, 2)) * exp(-a * pow(h_time[it] - t0, 2));
        h_wavelet[it] *= dt2dx2;
    }

    printf("SOURCE:\n");
    printf("\t%f\t:f0\n", f0);
    printf("\t%f\t:t0\n", t0);
    printf("\t%i\t:param.srcPosY - ox\n", param.srcPosY);
    printf("\t%i\t:param.srcPosX - oy\n", param.srcPosX);
    printf("\t%e\t:dt2dx2\n", dt2dx2);
    printf("\t%f\t:min wavelength [m]\n",(float)h_model.maxVel / (2*f0));
    printf("\t%f\t:ppw\n",(float)h_model.maxVel / (2*f0) / param.modelDx);

    // Set Output files

    int dimensions[3] = {nt,param.nReceptors,param.nShots};
    float spacings[3] = {1,1,1};
    int origins[3] = {0,0,0};
    sf_file Fdata_directWave = createFile3D("comOD",dimensions,spacings,origins);
    sf_file Fonly_directWave = createFile3D("OD",dimensions,spacings,origins);
    sf_file Fdata = createFile3D("data",dimensions,spacings,origins);

    // ===================MODELING======================
    modeling(param.modelNx, param.modelNy, param.taperBorder, param.nReceptors, nt, param.firstReceptorPos, param.lastReceptorPos, param.srcPosY, param.srcPosX, param.modelDx, param.modelDy, dt, h_model.extVelField, h_model.firstLayerVelField, h_tapermask, h_data, h_directwave,  h_wavelet, false, param.nShots, param.incShots, Fonly_directWave, Fdata_directWave, Fdata);
    // =================================================


    printf("Clean memory...");
    delete[] h_model.velField;
    delete[] h_model.extVelField;
    delete[] h_model.firstLayerVelField;
    delete[] h_data;
    delete[] h_directwave;
    delete[] h_abc;
    delete[] h_tapermask;
    delete[] h_time;
    delete[] h_wavelet;


    return 0;
}
