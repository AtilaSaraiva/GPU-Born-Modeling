#include "hip/hip_runtime.h"
/*
Hello world of wave propagation in CUDA. FDTD acoustic wave propagation in homogeneous medium. Second order accurate in time and eigth in space.

Oleg Ovcharenko
Vladimir Kazei, 2019

oleg.ovcharenko@kaust.edu.sa
vladimir.kazei@kaust.edu.sa
*/

#include <rsf.hh>
#include <iostream>
#include <string>
#include "stdio.h"
#include "math.h"
#include "stdlib.h"
#include "string.h"
/*
Add this to c_cpp_properties.json if linting isn't working for CUDA libraries
"includePath": [
                "/usr/local/cuda-10.0/targets/x86_64-linux/include",
                "${workspaceFolder}/**"
            ],
*/

#include "cuwaveprop2d.cu"

using namespace std;

void modeling(int nx, int ny, int nb, int nr, int nt, int gxbeg, int gxend, int isrc, int jsrc, float dx, float dy, float dt, float *h_vpe, float *h_dvpe, float *h_tapermask, float *h_data, float *h_directwave, float * h_wavelet, bool snaps, int nshots, int incShots, sf_file Fonly_directWave, sf_file Fdata_directWave, sf_file Fdata);

void dummyVelField(int nxb, int nyb, int nb, float *h_vpe, float *h_dvpe)
{
    for (int i = 0; i < nyb; i++){
        for (int j = 0; j < nxb; j++){
            h_dvpe[j * nyb + i]  = h_vpe[j * nyb + nb];
        }
    }
}

void expand(int nb, int nyb, int nxb, int nz, int nx, float *a, float *b)
/*< expand domain of 'a' to 'b':  a, size=nz*nx; b, size=nyb*nxb;  >*/
{
    int iz,ix;
    for     (ix=0;ix<nx;ix++) {
        for (iz=0;iz<nz;iz++) {
            b[(nb+ix)*nyb+(nb+iz)] = a[ix*nz+iz];
        }
    }
    for     (ix=0; ix<nxb; ix++) {
        for (iz=0; iz<nb; iz++)         b[ix*nyb+iz] = b[ix*nyb+nb];//top
        for (iz=nz+nb; iz<nyb; iz++) b[ix*nyb+iz] = b[ix*nyb+nb+nz-1];//bottom
    }

    for (iz=0; iz<nyb; iz++){
        for(ix=0; ix<nb; ix++)  b[ix*nyb+iz] = b[nb*nyb+iz];//left
        for(ix=nb+nx; ix<nxb; ix++)     b[ix*nyb+iz] = b[(nb+nx-1)*nyb+iz];//right
    }
}

void abc_coef (int nb, float *abc)
{
    for(int i=0; i<nb; i++){
        abc[i] = exp (-pow(0.001 * (nb - i + 1),2.0));
    }
}

void taper (int nx, int ny, int nb, float *abc, float *campo)
{
    int nxb = nx + 2 * nb;
    int nyb = ny + 2 * nb;
    for(int j=0; j<nxb; j++){
        for(int i=0; i<nb; i++){
            campo[j * nyb + i] *= abc[i];
            campo[j * nyb + (nb + ny + i)] *= abc[nb - i - 1];
        }
    }
    for(int i=0; i<nyb; i++){
        for(int j=0; j<nb; j++){
            campo[j * nyb + i] *= abc[j];
            campo[(nb + nx + j) * nyb + i] *= abc[nb - j - 1];
        }
    }
}

sf_file createFile3D (const char *name, int dimensions[3], float spacings[3], int origins[3])
{
    sf_file Fdata = NULL;
    Fdata = sf_output(name);
    char key_n[6],key_d[6],key_o[6];
    for (int i = 0; i < 3; i++){
        sprintf(key_n,"n%i",i+1);
        sprintf(key_d,"d%i",i+1);
        sprintf(key_o,"o%i",i+1);
        sf_putint(Fdata,key_n,dimensions[i]);
        sf_putint(Fdata,key_d,spacings[i]);
        sf_putint(Fdata,key_o,origins[i]);
    }

    return Fdata;
}

typedef struct{
    int nShots;
    int srcPosX;
    int srcPosY;
    int firstReceptorPos;
    int nReceptors;
    int lastReceptorPos;
    int incShots;
    int modelNx;
    int modelNy;
    int modelDx;
    int modelDy;
    int taperBorder;
} geometry;

geometry getParameters(sf_file FvelModel)
{
    geometry param;
    sf_getint("nr",&param.nReceptors);
    sf_getint("isrc",&param.srcPosY);
    sf_getint("jsrc",&param.srcPosX);
    sf_getint("gxbeg",&param.firstReceptorPos);
    sf_getint("nshots",&param.nShots);
    sf_getint("incShots",&param.incShots);
    sf_histint(FvelModel, "n1",&param.modelNy);
    sf_histint(FvelModel, "n2", &param.modelNx);
    param.lastReceptorPos = param.firstReceptorPos + param.nReceptors;
    param.taperBorder = 0.2 * param.modelNx;
    return param;
}

void test_getParameters (geometry parameters)
{
    cerr<<"param.incShots: "<<param.incShots<<endl;
    cerr<<"param.modelDims[0] "<<param.modelNx<<param.modelNy<<endl;
    cerr<<"param.nShots "<<param.nShots<<endl;
    cerr<<"param.nReceptors "<<param.nReceptors<<endl;
    cerr<<"param.firstReceptorPos "<<param.firstReceptorPos<<endl;
    cerr<<"param.lastReceptorPos "<<param.lastReceptorPos<<endl;
}


/*
===================================================================================
MAIN
===================================================================================
*/
int main(int argc, char *argv[])
{
    /* Main program that reads and writes data and read input variables */
    bool verb;
    sf_init(argc,argv); // init RSF
    if(! sf_getbool("verb",&verb)) verb=0;

    // Setting up I/O files
    sf_file Fvel=NULL;
    Fvel = sf_input("vel");

    // Getting command line parameters
    int nr; sf_getint("nr",&nr);
    int isrc; sf_getint("isrc",&isrc);
    int jsrc; sf_getint("jsrc",&jsrc);
    int gxbeg; sf_getint("gxbeg",&gxbeg);
    int nshots; sf_getint("nshots",&nshots);
    int incShots; sf_getint("incShots",&incShots);
    int gxend = gxbeg + nr;

    geometry parameters = getParameters(Fvel);

    // R/W axes
    sf_axis ax,ay;
    int nx, ny, nb, nxb, nyb;
    float dx, dy;
    ay = sf_iaxa(Fvel,1); ny = sf_n(ay); dy = sf_d(ay);
    ax = sf_iaxa(Fvel,2); nx = sf_n(ax); dx = sf_d(ax);
    cerr<<"nx = "<<nx<<endl;
    cerr<<"ny = "<<ny<<endl;

    size_t nxy = nx * ny;
    nb = 0.2 * nx;
    nxb = nx + 2 * nb;
    nyb = ny + 2 * nb;
    size_t nbxy = nxb * nyb;
    size_t nbytes = nbxy * sizeof(float);/* bytes to store nx * ny */

    // Allocate memory for velocity model
    float *h_vp = new float[nxy]; sf_floatread(h_vp, nxy, Fvel);
    float *h_vpe = new float[nbxy];
    memset(h_vpe,0,nbytes);
    //extendVelField(nx, ny, nb, h_vp, h_vpe);
    expand(nb, nyb, nxb, ny, nx, h_vp, h_vpe);
    float _vp = h_vp[0];
    for(int i=1; i < nxy; i++){
        if(h_vp[i] > _vp){
            _vp = h_vp[i];
        }
    }

    printf("MODEL:\n");
    printf("\t%i x %i\t:ny x nx\n", ny, nx);
    printf("\t%f\t:dx\n", dx);
    printf("\t%f\t:h_vp[0]\n", h_vp[0]);


    // Allocate memory for dummy velocity model and seismogram
    float *h_dvpe = new float[nbxy];
    dummyVelField(nxb, nyb, nb, h_vpe, h_dvpe);

    cerr<<"vp = "<<_vp<<endl;
    cerr<<"nb = "<<nb<<endl;

    // Taper mask
    float *h_abc = new float[nb];
    float *h_tapermask = new float[nbxy];
    for(int i=0; i < nbxy; i++){
        h_tapermask[i] = 1;
    }
    abc_coef(nb, h_abc);
    taper(nx, ny, nb, h_abc, h_tapermask);


    // Time stepping
    float t_total = 2.5;               /* total time of wave propagation, sec */
    float dt = 0.5 * dx / _vp;         /* time step assuming constant vp, sec */
    int nt = round(t_total / dt);      /* number of time steps */
    int snap_step = round(0.1 * nt);   /* save snapshot every ... steps */

    printf("TIME STEPPING:\n");
    printf("\t%e\t:t_total\n", t_total);
    printf("\t%e\t:dt\n", dt);
    printf("\t%i\t:nt\n", nt);

    // Data
    float *h_data = new float[nr * nt];
    float *h_directwave = new float[nr * nt];

    // Source
    float f0 = 10.0;                    /* source dominant frequency, Hz */
    float t0 = 1.2 / f0;                /* source padding to move wavelet from left of zero */

    float *h_wavelet, *h_time;
    float tbytes = nt * sizeof(float);
    h_time = (float *)malloc(tbytes);
    h_wavelet = (float *)malloc(tbytes);

    // Fill source waveform vector
    float a = PI * PI * f0 * f0;            /* const for wavelet */
    float dt2dx2 = (dt * dt) / (dx * dx);   /* const for fd stencil */
    for (int it = 0; it < nt; it++)
    {
        h_time[it] = it * dt;
        // Ricker wavelet (Mexican hat), second derivative of Gaussian
        h_wavelet[it] = 1e10 * (1.0 - 2.0 * a * pow(h_time[it] - t0, 2)) * exp(-a * pow(h_time[it] - t0, 2));
        h_wavelet[it] *= dt2dx2;
    }

    printf("SOURCE:\n");
    printf("\t%f\t:f0\n", f0);
    printf("\t%f\t:t0\n", t0);
    printf("\t%i\t:isrc - ox\n", isrc);
    printf("\t%i\t:jsrc - oy\n", jsrc);
    printf("\t%e\t:dt2dx2\n", dt2dx2);
    printf("\t%f\t:min wavelength [m]\n",(float)_vp / (2*f0));
    printf("\t%f\t:ppw\n",(float)_vp / (2*f0) / dx);

    // Set Output files

    int dimensions[3] = {nt,nr,nshots};
    float spacings[3] = {1,1,1};
    int origins[3] = {0,0,0};
    sf_file Fdata_directWave = createFile3D("comOD",dimensions,spacings,origins);
    sf_file Fonly_directWave = createFile3D("OD",dimensions,spacings,origins);
    sf_file Fdata = createFile3D("data",dimensions,spacings,origins);

    // ===================MODELING======================
    modeling(nx, ny, nb, nr, nt, gxbeg, gxend, isrc, jsrc, dx, dy, dt, h_vpe, h_dvpe, h_tapermask, h_data, h_directwave,  h_wavelet, false, nshots, incShots, Fonly_directWave, Fdata_directWave, Fdata);
    // =================================================


    printf("Clean memory...");
    delete[] h_vp;
    delete[] h_vpe;
    delete[] h_dvpe;
    delete[] h_data;
    delete[] h_directwave;
    delete[] h_abc;
    delete[] h_tapermask;
    delete[] h_time;
    delete[] h_wavelet;


    return 0;
}
