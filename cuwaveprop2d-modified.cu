#include "hip/hip_runtime.h"
/*
Hello world of wave propagation in CUDA. FDTD acoustic wave propagation in homogeneous medium. Second order accurate in time and eigth in space.

Oleg Ovcharenko
Vladimir Kazei, 2019

oleg.ovcharenko@kaust.edu.sa
vladimir.kazei@kaust.edu.sa
*/

#include <rsf.hh>
#include <iostream>
#include "stdio.h"
#include "math.h"
#include "stdlib.h"
#include "string.h"
/*
Add this to c_cpp_properties.json if linting isn't working for CUDA libraries
"includePath": [
                "/usr/local/cuda-10.0/targets/x86_64-linux/include",
                "${workspaceFolder}/**"
            ],
*/
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

using namespace std;

// Check error codes for CUDA functions
#define CHECK(call)                                                \
    {                                                              \
        hipError_t error = call;                                  \
        if (error != hipSuccess)                                  \
        {                                                          \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
        }                                                          \
    }

#define PI 3.14159265359

// Padding for FD scheme
#define HALO 4
#define HALO2 8

// FD stencil coefficients
#define a0  -2.8472222f
#define a1   1.6000000f
#define a2  -0.2000000f
#define a3   0.0253968f
#define a4  -0.0017857f

// Block dimensions
#define BDIMX 32
#define BDIMY 32

// Shared memory tile dimenstions
#define SDIMX BDIMX + HALO2
#define SDIMY BDIMY + HALO2

// Constant device memory
__constant__ float c_coef[5]; /* coefficients for 8th order fd */
__constant__ int c_isrc;      /* source location, ox */
__constant__ int c_jsrc;      /* source location, oz */
__constant__ int c_nx;        /* x dim */
__constant__ int c_ny;        /* y dim */
__constant__ int c_nt;        /* time steps */
__constant__ float c_dt2dx2;  /* dt2 / dx2 for fd*/


// Extend the velocity field

void extendVelField(int nx, int ny, int nb, float *h_vp, float *h_vpe)
{
    int nxb = nx + 2 * nb;
    for(int j=nb; j<(nx+nb); j++){
        for(int i=0; i<nb; i++){
            h_vpe[i * nxb + j] = h_vp[j-nb];
            h_vpe[(nb + ny + i) * nxb + j] = h_vp[(ny - 1) * nx + (j - nb)];
        }
    }
    for(int i=nb; i<(ny+nb); i++){
        for(int j=0; j<nb; j++){
            h_vpe[i * nxb + j] = h_vp[(i - nb) * nx];
            h_vpe[i * nxb + nb + nx + j] = h_vp[(i - nb) * nx + (nx - 1)];
        }
    }
    for(int j=0; j<nb; j++){
        for(int i=0; i<nb; i++){
            h_vpe[i * nxb + j] = h_vp[0];
            h_vpe[i * nxb + (j + nx + nb)] = h_vp[nx - 1];
            h_vpe[(i + ny + nb) * nxb + j] = h_vp[(ny - 1) * nx];
            h_vpe[(i + ny + nb) * nxb + (j + nx + nb)] = h_vp[ny * nx - 1];
        }
    }
    for(int j=0; j<nx; j++){
        for(int i=0; i<ny; i++){
            h_vpe[(i + nb) * nxb + j + nb] = h_vp[i * nx + j];
        }
    }
}

void abc_coef (int nb, float *abc)
{
    for(int i=0; i<nb; i++){
        abc[i] = exp (-pow(0.008 * (nb - i + 1),2.0));
    }
}

void taper (int nx, int ny, int nb, float *abc, float *campo)
{
    int nxb = nx + 2 * nb;
    int nyb = ny + 2 * nb;
    for(int j=0; j<nxb; j++){
        for(int i=0; i<nb; i++){
            campo[i * nxb + j] *= abc[i];
            campo[(nb + ny + i) * nxb + j] *= abc[nb - i - 1];
        }
    }
    for(int i=0; i<nyb; i++){
        for(int j=0; j<nb; j++){
            campo[i * nxb + j] *= abc[j];
            campo[i * nxb + nb + nx + j] *= abc[nb - j - 1];
        }
    }
}


// Save snapshot as a binary, filename snap/snap_tag_it_ny_nx
void saveSnapshotIstep(int it, float *data, int nx, int ny, const char *tag)
{
    /*
    it      :timestep id
    data    :pointer to an array in device memory
    nx, ny  :model dimensions
    tag     :user-defined file identifier
    */

    // Array to store wavefield
    unsigned int isize = nx * ny * sizeof(float);
    float *iwave = (float *)malloc(isize);
    CHECK(hipMemcpy(iwave, data, isize, hipMemcpyDeviceToHost));

    char fname[32];
    sprintf(fname, "snap/snap_%s_%i_%i_%i", tag, it, ny, nx);

    FILE *fp_snap = fopen(fname, "w");

    fwrite(iwave, sizeof(float), nx * ny, fp_snap);
    printf("\tSave...%s: nx = %i ny = %i it = %i tag = %s\n", fname, nx, ny, it, tag);
    fflush(stdout);
    fclose(fp_snap);

    free(iwave);
    return;
}

// Add source wavelet
__global__ void kernel_add_wavelet(float *d_u, float *d_wavelet, int it)
{
    /*
    d_u             :pointer to an array on device where to add source term
    d_wavelet       :pointer to an array on device with source signature
    it              :time step id
    */
    unsigned int gx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int gy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = gy * c_nx + gx;

    if ((gx == c_isrc) && (gy == c_jsrc))
    {
        d_u[idx] += d_wavelet[it];
    }
}

__device__ void set_halo(float *global, float shared[][SDIMX], int tx, int ty, int sx, int sy, int gx, int gy, int nx, int ny)
{
    /*
    global      :pointer to an array in global memory (gmem)
    shared      :2D array in shared device memory
    tx, ty      :thread id's in a block
    sx, sy      :thread id's in a shared memory tile
    gx, gy      :thread id's in the entire computational domain
    */

    // Each thread copies one value from gmem into smem
    shared[sy][sx] = global[gy * nx + gx];

    // Populate halo regions in smem for left, right, top and bottom boundaries of a block
    // if thread near LEFT border of a block
    if (tx < HALO)
    {
        // if global left
        if (gx < HALO)
        {
            // reflective boundary
            shared[sy][sx - HALO] = 0.0;
        }
        else
        {
            // if block left
            shared[sy][sx - HALO] = global[gy * nx + gx - HALO];
        }
    }
    // if thread near RIGHT border of a block
    if ((tx >= (BDIMX - HALO)) || ((gx + HALO) >= nx))
    {
        // if global right
        if ((gx + HALO) >= nx)
        {
            // reflective boundary
            shared[sy][sx + HALO] = 0.0;
        }
        else
        {
            // if block right
            shared[sy][sx + HALO] = global[gy * nx + gx + HALO];
        }
    }

    // if thread near BOTTOM border of a block
    if (ty < HALO)
    {
        // if global bottom
        if (gy < HALO)
        {
            // reflective boundary
            shared[sy - HALO][sx] = 0.0;
        }
        else
        {
            // if block bottom
            shared[sy - HALO][sx] = global[(gy - HALO) * nx + gx];
        }
    }

    // if thread near TOP border of a block
    if ((ty >= (BDIMY - HALO)) || ((gy + HALO) >= ny))
    {
        // if global top
        if ((gy + HALO) >= ny)
        {
            // reflective boundary
            shared[sy + HALO][sx] = 0.0;
        }
        else
        {
            // if block top
            shared[sy + HALO][sx] = global[(gy + HALO) * nx + gx];
        }
    }
}

// FD kernel
__global__ void kernel_2dfd(float *d_u1, float *d_u2, float *d_vp)
{
    // save model dims in registers as they are much faster
    const int nx = c_nx;
    const int ny = c_ny;

    // FD coefficient dt2 / dx2
    const float dt2dx2 = c_dt2dx2;

    // Thread address (ty, tx) in a block
    const unsigned int tx = threadIdx.x;
    const unsigned int ty = threadIdx.y;

    // Thread address (sy, sx) in shared memory
    const unsigned int sx = threadIdx.x + HALO;
    const unsigned int sy = threadIdx.y + HALO;

    // Thread address (gy, gx) in global memory
    const unsigned int gx = blockIdx.x * blockDim.x + tx;
    const unsigned int gy = blockIdx.y * blockDim.y + ty;

    // Global linear index
    const unsigned int idx = gy * nx + gx;

    // Allocate shared memory for a block (smem)
    __shared__ float s_u1[SDIMY][SDIMX];
    __shared__ float s_u2[SDIMY][SDIMX];
    __shared__ float s_vp[SDIMY][SDIMX];

    // If thread points into the physical domain
    if ((gx < nx) && (gy < ny))
    {
        // Copy regions from gmem into smem
        //       gmem, smem,  block, shared, global, dims
        set_halo(d_u1, s_u1, tx, ty, sx, sy, gx, gy, nx, ny);
        set_halo(d_u2, s_u2, tx, ty, sx, sy, gx, gy, nx, ny);
        set_halo(d_vp, s_vp, tx, ty, sx, sy, gx, gy, nx, ny);
        __syncthreads();

        // Central point of fd stencil, o o o o x o o o o
        float du2_xx = c_coef[0] * s_u2[sy][sx];
        float du2_yy = c_coef[0] * s_u2[sy][sx];

#pragma unroll
        for (int d = 1; d <= 4; d++)
        {
            du2_xx += c_coef[d] * (s_u2[sy][sx - d] + s_u2[sy][sx + d]);
            du2_yy += c_coef[d] * (s_u2[sy - d][sx] + s_u2[sy + d][sx]);
        }
        // Second order wave equation
        d_u1[idx] = 2.0 * s_u2[sy][sx] - s_u1[sy][sx] + s_vp[sy][sx] * s_vp[sy][sx] * (du2_xx + du2_yy) * dt2dx2;

        __syncthreads();
    }
}



/*
===================================================================================
MAIN
===================================================================================
*/
int main(int argc, char *argv[])
{
    /* Main program that reads and writes data and read input variables */
    bool verb;
    sf_init(argc,argv); // init RSF
    if(! sf_getbool("verb",&verb)) verb=0;

    // Setting up I/O files
    sf_file Fvel=NULL;
    Fvel = sf_input("vel");

    // R/W axes
    sf_axis ax,ay;
    int nx, ny, nb, nxb, nyb;
    float dx, dy;
    ay = sf_iaxa(Fvel,2); ny = sf_n(ay); dy = sf_d(ay);
    ax = sf_iaxa(Fvel,1); nx = sf_n(ax); dx = sf_d(ax);

    size_t nxy = nx * ny;
    nb = 0.2 * nx;
    nxb = nx + 2 * nb;
    nyb = ny + 2 * nb;
    size_t nbxy = nxb * nyb;
    size_t nbytes = nbxy * sizeof(float);/* bytes to store nx * ny */

    // Allocate memory for velocity model
    float *h_vp = new float[nxy]; sf_floatread(h_vp, nxy, Fvel);
    float *h_vpe = new float[nbxy];
    extendVelField(nx, ny, nb, h_vp, h_vpe);
    float _vp = h_vp[0];
    for(int i=1; i < nxy; i++){
        if(h_vp[i] > _vp){
            _vp = h_vp[i];
        }
    }

    cerr<<"vp = "<<_vp<<endl;
    cerr<<"nb = "<<nb<<endl;


    float *h_abc = new float[nb];
    abc_coef(nb, h_abc);
    taper(nx, ny, nb, h_abc, h_vpe);

    sf_file Fout=NULL;
    Fout = sf_output("oi");
    sf_putint(Fout,"n1",nxb);
    sf_putint(Fout,"n2",nyb);
    sf_floatwrite(h_vpe, nbxy, Fout);

    printf("MODEL:\n");
    printf("\t%i x %i\t:ny x nx\n", ny, nx);
    printf("\t%f\t:dx\n", dx);
    printf("\t%f\t:h_vp[0]\n", h_vp[0]);

    // Time stepping
    float t_total = 1.5;               /* total time of wave propagation, sec */
    float dt = 0.5 * dx / _vp;         /* time step assuming constant vp, sec */
    int nt = round(t_total / dt);      /* number of time steps */
    int snap_step = round(0.1 * nt);   /* save snapshot every ... steps */

    printf("TIME STEPPING:\n");
    printf("\t%e\t:t_total\n", t_total);
    printf("\t%e\t:dt\n", dt);
    printf("\t%i\t:nt\n", nt);

    // Source
    float f0 = 10.0;                    /* source dominant frequency, Hz */
    float t0 = 1.2 / f0;                /* source padding to move wavelet from left of zero */
    int isrc = round((float)nx / 2);    /* source location, ox */
    int jsrc = round((float)ny / 2);    /* source location, oz */

    float *h_wavelet, *h_time;
    float tbytes = nt * sizeof(float);
    h_time = (float *)malloc(tbytes);
    h_wavelet = (float *)malloc(tbytes);

    // Fill source waveform vector
    float a = PI * PI * f0 * f0;            /* const for wavelet */
    float dt2dx2 = (dt * dt) / (dx * dx);   /* const for fd stencil */
    for (int it = 0; it < nt; it++)
    {
        h_time[it] = it * dt;
        // Ricker wavelet (Mexican hat), second derivative of Gaussian
        h_wavelet[it] = 1e10 * (1.0 - 2.0 * a * pow(h_time[it] - t0, 2)) * exp(-a * pow(h_time[it] - t0, 2));
        h_wavelet[it] *= dt2dx2;
    }

    printf("SOURCE:\n");
    printf("\t%f\t:f0\n", f0);
    printf("\t%f\t:t0\n", t0);
    printf("\t%i\t:isrc - ox\n", isrc);
    printf("\t%i\t:jsrc - oy\n", jsrc);
    printf("\t%e\t:dt2dx2\n", dt2dx2);
    printf("\t%f\t:min wavelength [m]\n",(float)_vp / (2*f0));
    printf("\t%f\t:ppw\n",(float)_vp / (2*f0) / dx);

    // Allocate memory on device
    printf("Allocate and copy memory on the device...\n");
    float *d_u1, *d_u2, *d_vp, *d_wavelet, *d_abc;
    CHECK(hipMalloc((void **)&d_u1, nbytes))       /* wavefield at t-2 */
    CHECK(hipMalloc((void **)&d_u2, nbytes))       /* wavefield at t-1 */
    CHECK(hipMalloc((void **)&d_vp, nbytes))       /* velocity model */
    CHECK(hipMalloc((void **)&d_wavelet, tbytes)); /* source term for each time step */
    CHECK(hipMalloc((void **)&d_abc, nb * sizeof(float)));
    // Fill allocated memory with a value
    CHECK(hipMemset(d_u1, 0, nbytes))
    CHECK(hipMemset(d_u2, 0, nbytes))

    // Copy arrays from host to device
    CHECK(hipMemcpy(d_vp, h_vpe, nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_abc, h_abc, nb * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_wavelet, h_wavelet, tbytes, hipMemcpyHostToDevice));

    // Copy constants to device constant memory
    float coef[] = {a0, a1, a2, a3, a4};
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_coef), coef, 5 * sizeof(float)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_isrc), &isrc, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_jsrc), &jsrc, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_nx), &nx, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_ny), &ny, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_nt), &nt, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_dt2dx2), &dt2dx2, sizeof(float)));
    printf("\t%f MB\n", (4 * nbytes + tbytes)/1024/1024);
    printf("OK\n");

    // Print out specs of the main GPU
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, 0));
    printf("GPU0:\t%s\t%d.%d:\n", deviceProp.name, deviceProp.major, deviceProp.minor);
    printf("\t%lu GB:\t total Global memory (gmem)\n", deviceProp.totalGlobalMem / 1024 / 1024 / 1000);
    printf("\t%lu MB:\t total Constant memory (cmem)\n", deviceProp.totalConstMem / 1024);
    printf("\t%lu MB:\t total Shared memory per block (smem)\n", deviceProp.sharedMemPerBlock / 1024);
    printf("\t%d:\t total threads per block\n", deviceProp.maxThreadsPerBlock);
    printf("\t%d:\t total registers per block\n", deviceProp.regsPerBlock);
    printf("\t%d:\t warp size\n", deviceProp.warpSize);
    printf("\t%d x %d x %d:\t max dims of block\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("\t%d x %d x %d:\t max dims of grid\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    CHECK(hipSetDevice(0));

    // Print out CUDA domain partitioning info
    printf("CUDA:\n");
    printf("\t%i x %i\t:block dim\n", BDIMY, BDIMX);
    printf("\t%i x %i\t:shared dim\n", SDIMY, SDIMX);
    printf("CFL:\n");
    printf("\t%f\n", _vp * dt / dx);

    // Setup CUDA run
    dim3 block(BDIMX, BDIMY);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);


    // MAIN LOOP
    printf("Time loop...\n");
    for (int it = 0; it < nt; it++)
    {
        // These kernels are in the same stream so they will be executed one by one
        kernel_add_wavelet<<<grid, block>>>(d_u2, d_wavelet, it);
        kernel_2dfd<<<grid, block>>>(d_u1, d_u2, d_vp);
        CHECK(hipDeviceSynchronize());

        // Exchange time steps
        float *d_u3 = d_u1;
        d_u1 = d_u2;
        d_u2 = d_u3;

        // Save snapshot every snap_step iterations
        if ((it % snap_step == 0))
        {
            printf("%i/%i\n", it+1, nt);
            saveSnapshotIstep(it, d_u3, nx, ny,"u3");
        }
    }
    printf("OK\n");

    CHECK(hipGetLastError());

    printf("Clean memory...");
    delete[] h_vp;
    delete[] h_time;
    delete[] h_wavelet;

    CHECK(hipFree(d_u1));
    CHECK(hipFree(d_u2));
    CHECK(hipFree(d_vp));
    CHECK(hipFree(d_wavelet));
    printf("OK\n");
    CHECK(hipDeviceReset());

    return 0;
}
